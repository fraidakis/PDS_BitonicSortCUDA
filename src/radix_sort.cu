#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>  // NVIDIA's CUB library for high-performance GPU primitives
#include <time.h>

// Uncomment to enable verification against sequential sort
// #define VERIFY

void sequential_sort_verify(int *array, int *sequential_array, size_t size);

// Comparison function for qsort used in verification
// Returns difference between integers for ascending order sort
int compareAscending(const void *a, const void *b)
{
    return (*(int *)a - *(int *)b);
}

// Main radix sort function using NVIDIA's CUB library
void cubRadixSort(int *h_array, size_t size) {
    // Allocate device memory for input and output arrays
    int *d_keys_in;   // Input array on GPU
    int *d_keys_out;  // Output array on GPU
    hipMalloc(&d_keys_in, size * sizeof(int));
    hipMalloc(&d_keys_out, size * sizeof(int));

    // Create CUDA events for timing measurement
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Start timing
    hipEventRecord(start);

    // Transfer input array from host to device
    hipMemcpy(d_keys_in, h_array, size * sizeof(int), hipMemcpyHostToDevice);

    // CUB sorting preparation
    void *d_temp_storage = nullptr;
    size_t temp_storage_bytes = 0;

    // First call to determine required temporary storage size
    // This is a CUB requirement - we need to query the size first
    hipcub::DeviceRadixSort::SortKeys(nullptr, temp_storage_bytes, 
                                  d_keys_in, d_keys_out, size);

    // Allocate temporary storage required by CUB
    hipMalloc(&d_temp_storage, temp_storage_bytes);

    // Perform the actual radix sort
    // This sorts the array in-place on the GPU
    hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, 
                                  d_keys_in, d_keys_out, size);

    // Transfer sorted array back to host
    hipMemcpy(h_array, d_keys_out, size * sizeof(int), hipMemcpyDeviceToHost);

    // Stop timing and calculate duration
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("%f ms\n", milliseconds);

    // Cleanup GPU resources
    hipFree(d_keys_in);
    hipFree(d_keys_out);
    hipFree(d_temp_storage);
}

int main(int argc, char **argv)
{
    // Verify the command-line arguments (should be exactly 2: program name and q)
    if (argc != 2)
    {
        fprintf(stderr, "Usage: %s <q>\n", argv[0]);
        return 1;
    }

    // Convert argument to integer: q is the log2 of the number of elements to sort
    int q = atoi(argv[1]);

    // Calculate the total number of elements to sort (2^q)
    size_t size = 1 << q; // Using bit shifting to compute power of 2

    // Replace regular malloc with pinned memory allocation
    int *array;
    hipHostAlloc((void**)&array, size * sizeof(int), hipHostMallocDefault);


    // Seed the random number generator using the current time for varied results
    srand(time(NULL));

    // Fill the array with random integers (0 to 999)
    for (int i = 0; i < size; i++)
    {
        array[i] = rand() % 1000; // Random integer between 0 and 999
    }

#ifdef VERIFY
    // Use pinned memory for verification array as well
    int *sequential_array;
    hipHostAlloc((void**)&sequential_array, size * sizeof(int), hipHostMallocDefault);
    memcpy(sequential_array, array, size * sizeof(int));
#endif

    // Run the bitonic sort on the array
    cubRadixSort(array, size);

#ifdef VERIFY
    // Verify that the sorted array matches what a sequential sort produces
    sequential_sort_verify(array, sequential_array, size);
#endif

    // Replace free with hipHostFree
    hipHostFree(array);
#ifdef VERIFY
    hipHostFree(sequential_array);
#endif

    return 0;
}

// Function to verify the correctness of the bitonic sort by comparing
// with the result of the C standard library's sequential qsort function.
void sequential_sort_verify(int *array, int *sequential_array, size_t size)
{
    // Sort using qsort on a copy of the array. This is our reference.
    qsort(sequential_array, size, sizeof(int), compareAscending);

    // Compare each element of the two arrays. Report a mismatch if found.
    bool is_sorted = true;
    for (int i = 0; i < size; i++)
    {
        if (array[i] != sequential_array[i])
        {
            printf("Error: Mismatch at index %d: %d != %d\n", i, array[i], sequential_array[i]);
            is_sorted = false;
            break;
        }
    }

    printf("\n%s sorting %zu elements\n\n\n", is_sorted ? "SUCCESSFUL" : "FAILED", size);
}